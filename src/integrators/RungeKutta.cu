#include "hip/hip_runtime.h"
#define INLINE __device__
#include"physics/mass.h"
#include"physics/geopotential.h"
#include"physics/ring.h"
#include"physics/mass.impl"
#include"utils/logger.h"
#include<stdlib.h>
#include<algorithm>
#include<hip/hip_runtime.h>
#include<hip/hip_cooperative_groups.h>
#include<mutex>

//Maximum CUDA_CORES usage, can be adjusted according to physical cuda cores of GPU device.
//  Note, however, adjusting this can affect resulting ephemeris,
//      since the force superposition are rearranged, and some floating-point additions are reordered.
//  This impact should be on the order of floating-point round-off error,
//      but the difference can build up over time, and lead to a drastically differenent ephemeris.
#define CUDA_CORES 1920

#define CUDA_IMPL

#define WARP_SIZE 32
#define MAXBLOCKS (CUDA_CORES/WARP_SIZE)

std::mutex cuda_mutex;

//geopotential data, mlist[first].gpmodel==second
typedef std::vector<std::pair<int_t,const geopotential*>> gpdata_t;
//ring data, mlist[first].ringmodel==second
typedef std::vector<std::pair<int_t,const ring*>> ringdata_t;

#if 0
#define mycudaMalloc hipMalloc
#define mycudaFree hipFree
#define mymalloc malloc
#define myfree myfree
#else
void mycudaMalloc(void *devPtr,size_t size){
    static void *mem=0;
    static size_t memsize=0;
    if(memsize<size){
        hipFree(mem);
        hipMalloc(&mem,size);
        memsize=size;
    }
    *(void**)devPtr=mem;
}
void *mymalloc(size_t size){
    static void *mem=0;
    static size_t memsize=0;
    if(memsize<size){
        free(mem);
        mem=malloc(size);
        memsize=size;
    }
    return mem;
}
void myfree(void *){

}
void mycudaFree(void *){

}
#endif

struct cuda_rungekutta_kernel_config{
    //number of mass
    int nmass;
    int nblocks,mass_per_block;
    int nthreads,mass_per_thread;
    int_t n_step;
    fast_real dt;
    mass *dmlist;
    mass_state *x0,*f;
    real t_eph;

    void load(std::vector<mass> &mlist,gpdata_t &mgp,ringdata_t &mrg,fast_real _dt,int_t _nstep){
        dt=_dt;
        n_step=_nstep;
        int mn=mlist.size();
        nmass=mn;
        mass_per_block=(mn+MAXBLOCKS-1)/MAXBLOCKS;
        nblocks=(mn+mass_per_block-1)/mass_per_block;
        nthreads=WARP_SIZE*std::min(MAXBLOCKS/nblocks,(mn+WARP_SIZE-1)/WARP_SIZE);
        //Make sure nthreads is power of 2
        int new_nth;
        while(new_nth=nthreads&nthreads-1)nthreads=new_nth;
        mass_per_thread=(mn+nthreads-1)/nthreads;
        int_t grsize=0;
        for(int_t i=0;i<mn;++i){
            mass &m=mlist[i];
            if(m.gpmodel){
                int_t thissize=m.gpmodel->size();
                mgp.push_back({i,m.gpmodel});
                m.gpmodel=(geopotential*)grsize;
                grsize+=thissize;
            }
            if(m.ringmodel){
                int_t thissize=m.ringmodel->size();
                mrg.push_back({i,m.ringmodel});
                m.ringmodel=(ring*)grsize;
                grsize+=thissize;
            }
        }
        mycudaMalloc(&x0,nmass*(sizeof(mass)+26*sizeof(mass_state))+grsize);
        f=x0+nmass;
        dmlist=(mass*)(x0+26*nmass);
        void *grdata=mymalloc(grsize);
        for(auto &mgpi:mgp){
            int_t gpoffset=(int_t)mlist[mgpi.first].gpmodel;
            mlist[mgpi.first].gpmodel=(geopotential*)(gpoffset+(int_t)(dmlist+nmass));
            memcpy((geopotential*)(gpoffset+(int_t)grdata),mgpi.second,mgpi.second->size());
        }
        for(auto &mrgi:mrg){
            int_t rgoffset=(int_t)mlist[mrgi.first].ringmodel;
            mlist[mrgi.first].ringmodel=(ring*)(rgoffset+(int_t)(dmlist+nmass));
            memcpy((ring*)(rgoffset+(int_t)grdata),mrgi.second,mrgi.second->size());
        }
        hipMemcpy(dmlist,mlist.data(),nmass*sizeof(mass),hipMemcpyHostToDevice);
        hipMemcpy(dmlist+nmass,grdata,grsize,hipMemcpyHostToDevice);
        myfree(grdata);
    }

    void save(std::vector<mass> &mlist,gpdata_t &mgp,ringdata_t &mrg){
        hipMemcpy(mlist.data(),dmlist,nmass*sizeof(mass),hipMemcpyDeviceToHost);
        for(auto &mgpi:mgp){
            mlist[mgpi.first].gpmodel=mgpi.second;
        }
        for(auto &mrgi:mrg){
            mlist[mrgi.first].ringmodel=mrgi.second;
        }
        mycudaFree((mass_state*)dmlist-26*nmass);
    }
};

__constant__ cuda_rungekutta_kernel_config dkf;

struct maccel_1{
    fast_mpmat C_potential;
    fast_mpvec naccel;
    fast_real phi;
};
struct maccel_2{
    fast_mpvec gaccel,daccel,dtorque;
    fast_real min_distance;
    fast_real max_influence;
};

#include"physics/geopotential.impl"
#include"physics/ring.impl"

extern __shared__ char sharedMem[];
void __device__ accel_0(){//deform

    const fast_real c=CONSTANT_VALUE_C;
    const fast_real c2=c*c;

    int i0=blockIdx.x*dkf.mass_per_block;
    for(int di=0;di<dkf.mass_per_block;++di){
        int i=di+i0;
        if(i<dkf.nmass){
            mass &mi=dkf.dmlist[i];

            int_t max_iter=MAX_ANGULAR_VELOCITY_ITER;
            do{
                maccel_1 *tpmi=(maccel_1 *)sharedMem+threadIdx.x;
                tpmi[0].phi=0;
                tpmi[0].naccel=0;
                tpmi[0].C_potential=0;
                for(int dj=0;dj<dkf.nmass;dj+=blockDim.x){
                    int j=dj+threadIdx.x;
                    if(j<dkf.nmass&&i!=j){
                        mass &mj=dkf.dmlist[j];
                        DAMPED_TIDAL_DEFORMATION_MATRIX(tpmi[0]);
                    }
                }

                for(int wing=blockDim.x>>1;wing>1;wing>>=1){
                    __syncthreads();
                    if(threadIdx.x<wing){
                        tpmi[0].phi+=tpmi[wing].phi;
                        tpmi[0].naccel+=tpmi[wing].naccel;
                        tpmi[0].C_potential+=tpmi[wing].C_potential;
                    }
                }
                __syncthreads();
                if(threadIdx.x<1){
                    mi.phi=tpmi[0].phi+tpmi[1].phi;
                    mi.naccel=tpmi[0].naccel+tpmi[1].naccel;
                    mi.C_potential=tpmi[0].C_potential+tpmi[1].C_potential;
                }
                __syncthreads();

                __shared__ bool should_break;
                //angular accelerate
                if(threadIdx.x==0){
                    UPDATE_HARMONICS;
                    UPDATE_ANGULAR_VELOCITY;
                }
                __syncthreads();
                if(should_break)break;
            } while(--max_iter);
            if(threadIdx.x==0){
                PREPARE_RELATIVITY;
            }
        }
    }
}

void __device__ accel_1(){//accel
    const fast_real c=CONSTANT_VALUE_C;

    int i0=blockIdx.x*dkf.mass_per_block;
    for(int di=0;di<dkf.mass_per_block;++di){
        int i=di+i0;
        if(i<dkf.nmass){
            mass &mi=dkf.dmlist[i];
            maccel_2 *tpmi=(maccel_2*)sharedMem+threadIdx.x;
            tpmi[0].gaccel=0;
            tpmi[0].daccel=0;
            tpmi[0].dtorque=0;
            tpmi[0].min_distance=0;
            tpmi[0].max_influence=0;
            for(int dj=0;dj<dkf.nmass;dj+=blockDim.x){
                int j=dj+threadIdx.x;
                if(j<dkf.nmass&&i!=j){
                    mass &mj=dkf.dmlist[j];
                    RELATIVITY(tpmi[0]);
                    checked_maximize(tpmi[0].min_distance,rr);
                    checked_maximize(tpmi[0].max_influence,tp_dg);
                    //to avoid reduce cross thread block, re-calculate daccel instead of using anti-force
                    ROTATIONAL_TIDAL_DEFORMATION_NANTI_FORCE(tpmi[0]);
                    LENSE_THIRRING(tpmi[0]);
                    RADIATION_PRESSURE(tpmi[0]);

                    //higher harmonics will be done later
                }
            }

            for(int wing=blockDim.x>>1;wing>1;wing>>=1){
                __syncthreads();
                if(threadIdx.x<wing){
                    tpmi[0].gaccel+=tpmi[wing].gaccel;
                    tpmi[0].daccel+=tpmi[wing].daccel;
                    tpmi[0].dtorque+=tpmi[wing].dtorque;
                    checked_maximize(tpmi[0].min_distance,tpmi[wing].min_distance);
                    checked_maximize(tpmi[0].max_influence,tpmi[wing].max_influence);
                }
            }
            __syncthreads();
            if(threadIdx.x<1){
                mi.gaccel=tpmi[0].gaccel+tpmi[1].gaccel;
                mi.daccel=tpmi[0].daccel+tpmi[1].daccel;
                mi.dtorque=tpmi[0].dtorque+tpmi[1].dtorque;
                checked_maximize(tpmi[0].min_distance,tpmi[1].min_distance);
                checked_maximize(mi.min_distance,tpmi[0].min_distance);
                checked_maximize(tpmi[0].max_influence,tpmi[1].max_influence);
                checked_maximize(mi.max_influence,tpmi[0].max_influence);
            }
            __syncthreads();
        }
    }
}

void __device__ accel_2(){//higher harmonics
    int mn=dkf.nmass;
    int i0=blockIdx.x*dkf.mass_per_block;

    for(int j=0;j<dkf.nmass;++j){
        mass &mi=dkf.dmlist[j];
        if(mi.gpmodel||mi.ringmodel){
            maccel_2 *tpmi=(maccel_2*)sharedMem+threadIdx.x;
            tpmi[0].daccel=0;
            tpmi[0].dtorque=0;

            for(int di=0;di<dkf.mass_per_block;di+=blockDim.x){
                int i=i0+di+threadIdx.x;
                if(i<mn&&di+threadIdx.x<dkf.mass_per_block&&i!=j){
                    mass &mj=dkf.dmlist[i];
                    fast_mpvec an(0);
                    fast_mpvec r=mj.r-mi.r;
                    if(mi.gpmodel){
                        fast_mpmat fmis(mi.s);
                        fast_mpvec lr=fmis.tolocal(r);
                        an+=fmis.toworld(mi.gpmodel->cuda_sum(mi.R,lr));
                    }
                    if(mi.ringmodel){
                        fast_mpvec migl=mi.GL;
                        fast_mpmat fgls(migl.asc_node(),0,migl/migl.norm());
                        fast_mpvec lr=fgls.tolocal(r);
                        an+=fgls.toworld(mi.ringmodel->cuda_sum(mi.R,lr));
                    }

                    APPLY_NONPOINT_FORCE(tpmi[0]);
                }
            }
            //the initial wing should be lift to 2's power and the mask should be adjusted.
            for(int wing=blockDim.x>>1;wing>1;wing>>=1){
                __syncthreads();
                if(threadIdx.x<wing){
                    tpmi[0].daccel+=tpmi[wing].daccel;
                    tpmi[0].dtorque+=tpmi[wing].dtorque;
                }
            }
            __syncthreads();
            if(threadIdx.x<1){
                //Note: by construction, nblocks <= nmass
                //      so we store partial acceleration per block in global mlist memory
                mass &mbi=dkf.dmlist[blockIdx.x];
                mbi.idaccel=tpmi[0].daccel+tpmi[1].daccel;
                mbi.idtorque=tpmi[0].dtorque+tpmi[1].dtorque;
            }
            __syncthreads();


            cooperative_groups::grid_group grid=cooperative_groups::this_grid();
            grid.sync();
            if(blockIdx.x==0){
                // block 0
                tpmi[0].daccel=0;
                tpmi[0].dtorque=0;
                for(int i=threadIdx.x;i<gridDim.x;i+=blockDim.x){
                    mass &mj=dkf.dmlist[i];
                    tpmi[0].daccel+=mj.idaccel;
                    tpmi[0].dtorque+=mj.idtorque;
                }

                for(int wing=blockDim.x>>1;wing>1;wing>>=1){
                    __syncthreads();
                    if(threadIdx.x<wing){
                        tpmi[0].daccel+=tpmi[wing].daccel;
                        tpmi[0].dtorque+=tpmi[wing].dtorque;
                    }
                }
                __syncthreads();
                if(threadIdx.x<1){
                    mi.daccel+=tpmi[0].daccel+tpmi[1].daccel;
                    mi.dtorque+=tpmi[0].dtorque+tpmi[1].dtorque;
                }
                __syncthreads();

            }
            grid.sync();
        }
    }
}

void __device__ Cuda_accel(){
    const fast_real c=CONSTANT_VALUE_C;
    const fast_real c2=c*c;
    cooperative_groups::grid_group grid=cooperative_groups::this_grid();

    mass *x=dkf.dmlist;
    int mn=dkf.nmass;
    int i0=blockIdx.x*dkf.mass_per_block;

    grid.sync();
    accel_0();
    grid.sync();
    accel_1();
    grid.sync();
    accel_2();
    grid.sync();
    for(int di=0;di<dkf.mass_per_block;di+=blockDim.x){
        int i=i0+di+threadIdx.x;
        if(i<mn&&di+threadIdx.x<dkf.mass_per_block){
            mass &mi=x[i];
            FINALIZE_RELATIVITY;

            if(mi.ringmodel){
                RING_CORRECTION;
            }
        }
    }
}

#include"RungeKutta.impl"

static void Cuda_OnError(hipError_t cudaerr){
    if(cudaerr==hipSuccess)return;
    LogCritical(
        "\n\nCritical Error : [Cuda] %s\n\n"
        "    Try set CUDA_CORES to a less number, or get a better GPU.\n"
        "       (now CUDA_CORES = %d, defined in %s)\n"
        "    If you believe this is a bug, please contact author via github.\n"
        "    The program will now exit.\n"
        ,hipGetErrorString(cudaerr),CUDA_CORES,__FILE__);
    exit(-1);
}

void msystem::Cuda_RungeKutta12(fast_real dt,int_t n_step){
    if(n_step<=0)return;
    gpdata_t mgp;
    ringdata_t mrg;
    cuda_rungekutta_kernel_config kf;
    cuda_mutex.lock();
    kf.load(mlist,mgp,mrg,dt,n_step);
    kf.t_eph=t_eph;
    hipMemcpyToSymbol(HIP_SYMBOL(dkf),&kf,sizeof(kf));
    //Cuda_Kernel<<<kf.nblocks,kf.nthreads>>>();
    hipError_t cudaerr=hipLaunchCooperativeKernel(
        (void*)Cuda_RungeKutta_Kernel,
        dim3(kf.nblocks),
        dim3(kf.nthreads),
        nullptr,
        kf.nthreads*std::max(sizeof(maccel_1),sizeof(maccel_2))
    );
    hipDeviceSynchronize();
    kf.save(mlist,mgp,mrg);
    cuda_mutex.unlock();
    if(cudaerr!=hipSuccess)Cuda_OnError(cudaerr);
}

void __global__ Cuda_accel_Kernel(){
    Cuda_accel();
}

void msystem::Cuda_accel(){
    gpdata_t mgp;
    ringdata_t mrg;
    cuda_rungekutta_kernel_config kf;
    cuda_mutex.lock();
    kf.load(mlist,mgp,mrg,0,0);
    kf.t_eph=t_eph;
    hipMemcpyToSymbol(HIP_SYMBOL(dkf),&kf,sizeof(kf));
    //Cuda_Kernel<<<kf.nblocks,kf.nthreads>>>();
    hipError_t cudaerr=hipLaunchCooperativeKernel(
        (void*)Cuda_accel_Kernel,
        dim3(kf.nblocks),
        dim3(kf.nthreads),
        nullptr,
        kf.nthreads*std::max(sizeof(maccel_1),sizeof(maccel_2))
    );
    hipDeviceSynchronize();
    kf.save(mlist,mgp,mrg);
    cuda_mutex.unlock();
    if(cudaerr!=hipSuccess)Cuda_OnError(cudaerr);
}
